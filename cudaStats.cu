
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#define THREADS_PER_BLOCK 1024
using namespace std;

__global__ void subAvg(int* input,int count,int avg)
{
	int index = blockDim.x*blockIdx.x + threadIdx.x;
	if(index<count)
	input[index] = pow(input[index]-avg,2);
}

__global__ void max(int* input,int count)
{
	int blockStartPoint = blockDim.x*blockIdx.x;
	int threadWithinBlock = threadIdx.x;
	int scopeSize = 1;
	while(scopeSize<=THREADS_PER_BLOCK)
	{
		int threadLimit = THREADS_PER_BLOCK/scopeSize;
		if(threadWithinBlock<threadLimit)
		{
			int first = blockStartPoint*2 + threadWithinBlock*scopeSize*2;
			int second = first + scopeSize;
			if(first<count && second<count)
			{
				if(input[second]>input[first])
				input[first] = input[second];
			}
		}
		__syncthreads();
		scopeSize*=2;		
	}

}
__global__ void maxFinalize(int* input,int count)
{
	int maximum = input[0];
	for(int i=2048;i<count;i+=2048)
	{
		if(input[i]>maximum)
		maximum = input[i];
	}
	input[0] = maximum;
}

__global__ void min(int* input,int count)
{
	int blockStartPoint = blockDim.x*blockIdx.x;
	int threadWithinBlock = threadIdx.x;
	int scopeSize = 1;
	while(scopeSize<=THREADS_PER_BLOCK)
	{
		int threadLimit = THREADS_PER_BLOCK/scopeSize;
		if(threadWithinBlock<threadLimit)
		{
			int first = blockStartPoint*2 + threadWithinBlock*scopeSize*2;
			int second = first + scopeSize;
			if(first<count && second<count)
			{
				if(input[second]<input[first])
				input[first] = input[second];
			}
		}
		__syncthreads();
		scopeSize*=2;		
	}

}
__global__ void minFinalize(int* input,int count)
{
	int minimum = input[0];
	for(int i=2048;i<count;i+=2048)
	{
		if(input[i]<minimum)
		minimum = input[i];
	}
	input[0] = minimum;
}

__global__ void sum(int* input,int count)
{
	int blockStartPoint = blockDim.x*blockIdx.x;
	int threadWithinBlock = threadIdx.x;
	int scopeSize = 1;
	while(scopeSize<=THREADS_PER_BLOCK)
	{
		int threadLimit = THREADS_PER_BLOCK/scopeSize;
		if(threadWithinBlock<threadLimit)
		{
			int first = blockStartPoint*2 + threadWithinBlock*scopeSize*2;
			int second = first + scopeSize;
			if(first<count && second<count)
			{
				input[first] += input[second];
			}
		}
		__syncthreads();
		scopeSize*=2;		
	}

}

__global__ void sum2(int* input,int count)
{
	int blockStartPoint = blockDim.x*blockIdx.x;
	int threadWithinBlock = threadIdx.x;
	int scopeSize = 1;
	while(scopeSize<=THREADS_PER_BLOCK)
	{
		int threadLimit = THREADS_PER_BLOCK/scopeSize;
		if(threadWithinBlock<threadLimit)
		{
			int first = blockStartPoint*2 + threadWithinBlock*scopeSize*2;
			int second = first + scopeSize;
			if(first<count && second<count)
			{
				input[first] += input[second];
			}
		}
		__syncthreads();
		scopeSize*=2;		
	}

}
__global__ void sumUp(int* input,int count)
{
	for(int i=2048;i<count;i+=2048)
	{
		input[0] += input[i];
	}
}

int main(int argc, char const *argv[])
{	
	srand(3);
	//common part
	int count = 0;
	cout<<"Enter the number of elements:";
	cin>>count;
	int size = count * sizeof(int);
	int h[count];	     //allocating host memory
	int *d;
	hipMalloc(&d,size); //allocating device memory
	int blockSize = 1024;//initializing the max block size
	float numBlocks = floor((count+blockSize-1)/blockSize);
	numBlocks = ceil(numBlocks/2);//calculating number of blocks

	cout<<"Elements are:"<<endl;
	for (int i = 0; i < count; i++)
	{
		h[i] = i + 1;
		cout<<h[i]<<"\t";
	}
	
	//calculating minimum
	hipMemcpy(d,h,size,hipMemcpyHostToDevice);
	min<<<numBlocks,blockSize>>> (d,count);
	minFinalize<<<1,1>>> (d,count);
	int result;
	hipMemcpy(&result,d,sizeof(int),hipMemcpyDeviceToHost);
	cout<<"Minimum Element:"<<result<<endl;

	//calculating maximum
	hipMemcpy(d,h,size,hipMemcpyHostToDevice);
	max<<<numBlocks,blockSize>>> (d,count);
	maxFinalize<<<1,1>>> (d,count);
	hipMemcpy(&result,d,sizeof(int),hipMemcpyDeviceToHost);
	cout<<"Maximum Element:"<<result<<endl;

	//calculating sum
	hipMemcpy(d,h,size,hipMemcpyHostToDevice);
	sum<<<numBlocks,blockSize>>> (d,count);
	sumUp<<<1,1>>> (d,count);
	hipMemcpy(&result,d,sizeof(int),hipMemcpyDeviceToHost);
	cout<<"Sum is "<<result<<endl;
	cout<<"Correct sum(by formula)*ONLY IF INPUT IS 1...n* is:"<<count*(2+(count-1))/2<<endl;
	int sum = result;
	int average = sum/count;
	cout<<"Average is:"<<average<<endl;
	//calculating variance and standard deviation
	hipMemcpy(d,h,size,hipMemcpyHostToDevice);
	int subAvgnumBlocks = (count+blockSize-1)/blockSize;
	subAvg<<<subAvgnumBlocks,blockSize>>>(d,count,average);
	sum2<<<numBlocks,blockSize>>> (d,count);
	sumUp<<<1,1>>>(d,count);
	hipMemcpy(&result,d,sizeof(int),hipMemcpyDeviceToHost);
	cout<<"Variance is "<<result<<endl;
	cout<<"Standard Deviation is "<<sqrt(result)<<endl;




	getchar();
	hipFree(d);

	return 0;
}
